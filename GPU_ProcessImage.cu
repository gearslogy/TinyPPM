#include "hip/hip_runtime.h"
#include "GPU_ProcessImage.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#define THREADS_PER_BLOCK 1024

#define CHECK_CUDA_STATUS(STATUS)\
{\
    if (STATUS != hipSuccess)\
    {\
        fprintf(stdout,"Error in line %d\n ",__LINE__);\
    }\
}\

__global__ void gpuGreyImage(PPMIO::RGB *dev_srcImage , PPMIO::RGB *dev_desImage,int npixels)
{
	const long index = threadIdx.x + blockDim.x*blockIdx.x;
	if(index < npixels*3)
	{
		unsigned char r = unsigned char(float(dev_srcImage[index].R) * 0.244f);
		unsigned char g = unsigned char(float(dev_srcImage[index].G) * 0.587f);
		unsigned char b = unsigned char(float(dev_srcImage[index].B) * 0.114f);
		unsigned char i = r + g +b;
		dev_desImage[index].R = i;
		dev_desImage[index].G = i;
		dev_desImage[index].B = i;
	}
}

void processImage(PPMIO::RGB *host_srcImage, 
		PPMIO::RGB *host_desImage,int npixels)
{
	printf("Test in gpu compile\n");
	hipError_t cstat;
	cstat = hipSetDevice(0);
	CHECK_CUDA_STATUS(cstat);

	PPMIO::RGB *dev_src;
	PPMIO::RGB *dev_des;
	cstat = hipMallocManaged(&dev_src,sizeof(PPMIO::RGB) * npixels);
	CHECK_CUDA_STATUS(cstat);
	cstat = hipMallocManaged(&dev_des,sizeof(PPMIO::RGB) * npixels);
	CHECK_CUDA_STATUS(cstat);

	cstat = hipMemcpy(dev_src,host_srcImage,sizeof(PPMIO::RGB)*npixels,hipMemcpyHostToDevice);



	int NBLOCKS = npixels/THREADS_PER_BLOCK;
	if(NBLOCKS<=0)
	{
		NBLOCKS = 1;
	}
	gpuGreyImage<<<NBLOCKS,THREADS_PER_BLOCK>>>(dev_src,dev_des,npixels);

	cstat = hipMemcpy(host_desImage,dev_des,sizeof(PPMIO::RGB)*npixels,hipMemcpyDeviceToHost);
	CHECK_CUDA_STATUS(cstat);
	
	hipFree(dev_src);
	hipFree(dev_des);


}